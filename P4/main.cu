
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"
#include "string.h"
#include "math.h"

#define XRANGE 1
#define YRANGE 1

#define THRESH 1e-12

extern "C"
void VTK_out(const int N, const int M, const double *Xmin, const double *Xmax,
             const double *Ymin, const double *Ymax, const double *T,
             const int index);

//__device__ volatile int d_error_mutex = 0;
//__device__ double * d_conv_error;

__global__
void prepare_grids(double *T, double * T_tmp, double *S, double * errors, long * grid_size, long * internal_size, int * Px, int * Py) {
	long id = blockIdx.x*blockDim.x + threadIdx.x;
	long mapped_id = id-(2*(id/Px[0])-1)-(Px[0]);
	if(id >= grid_size[0])
		return;
	double val = (id%Px[0])*((double)XRANGE/Px[0])*powf(2.718281828, (id/Px[0])*((double)YRANGE/Py[0]));
	S[id] = val;
	if(id/Px[0]==0 || id/Px[0]==Py[0]-1 || id%Px[0]==0 || id%Px[0]==Px[0]-1) { 
		T[id] = val;
		T_tmp[id] = val;
		return;
	} else {
		T[id] = 0;
		T_tmp[id] = 0;
	}
	errors[mapped_id] = 0;
}

__global__
void update_temporary(double * T, double * T_tmp, double * S, double * errors, long * grid_size, int * Px, int * Py) {
	long id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=grid_size[0] || id/Px[0]==0 || id/Px[0]==Py[0]-1 || id%Px[0]==0 || id%Px[0]==Px[0]-1)
		return;
	T_tmp[id] = (-1*S[id]*powf((((double)XRANGE/Px[0])*((double)YRANGE/Py[0])),2)+(T[id-1]+T[id+1])*powf(((double)YRANGE/Py[0]),2)+ \
	(T[id-Px[0]]+T[id+Px[0]])*powf(((double)XRANGE/Px[0]),2))/(2*powf(((double)XRANGE/Px[0]),2)+2*powf(((double)YRANGE/Py[0]),2));
}

__global__
void update_real(double * T, double * T_tmp, double * S, double * errors, long * grid_size, int * Px, int * Py) {
	long id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=grid_size[0] || id/Px[0]==0 || id/Px[0]==Py[0]-1 || id%Px[0]==0 || id%Px[0]==Px[0]-1)
		return;
	T[id] = T_tmp[id];
}

__global__ 
void get_abs_error(double *T, double *S, long * grid_size, double * d_abs_error) {	
	long id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id >= grid_size[0])
		return;
	if(id==0)
		*d_abs_error = 0;
	__syncthreads();
	double val = fabs(T[id] - S[id]);
	val = powf(val,3);
	atomicAdd(d_abs_error, val);
}

__global__
void get_error(double *T, double *T_tmp, long * grid_size, int * Px, int * Py, double * d_conv_error) {
	long id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id >= grid_size[0])
		return;
	if(id==0)
		*d_conv_error = 0;
	__syncthreads();
	double val = fabs(T[id] - T_tmp[id]);
	val = powf(val,3);
	atomicAdd(d_conv_error, val);
}

int main(int argc, char **argv) {
	if(argc!=3) {
		printf("Usage:\n\t./main [xdim] [ydim]\n");
		exit(0);
	}
	int h_Px = atoi(argv[1]);
	int h_Py = atoi(argv[2]);
	long h_grid_size = h_Px*h_Py;
	long h_internal_size = h_grid_size - 2*h_Px - 2*h_Py + 4;
	int *d_Px, *d_Py;
	long *d_grid_size, *d_internal_size;
	hipMalloc((void**)&d_Px, sizeof(int));
	hipMalloc((void**)&d_Py, sizeof(int));
	hipMalloc((void**)&d_grid_size, sizeof(long));
	hipMalloc((void**)&d_internal_size, sizeof(long));
	hipMemcpy(d_Px, &h_Px, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Py, &h_Py, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_grid_size, &h_grid_size, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_internal_size, &h_internal_size, sizeof(long), hipMemcpyHostToDevice);


	// Allocate grids in Host to recieve from GPU (pinned memory)
	double * h_S;
	double * h_T;
	double * h_T_tmp;
	hipHostMalloc((void**)&h_S, h_grid_size*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void**)&h_T, h_grid_size*sizeof(double), hipHostMallocDefault);	
	hipHostMalloc((void**)&h_T_tmp, h_grid_size*sizeof(double), hipHostMallocDefault);

	// Allocate grids in GPU memory
	double * d_S;
	double * d_T;
	double * d_T_tmp;
	double * d_errors;
	hipMalloc((double**)&d_S, h_grid_size*sizeof(double));
	hipMalloc((double**)&d_T, h_grid_size*sizeof(double));	
	hipMalloc((double**)&d_T_tmp, h_grid_size*sizeof(double));
	hipMalloc((double**)&d_errors, h_internal_size*sizeof(double));
	double * d_conv_error;
	double * d_abs_error;
	hipMalloc((double**)&d_conv_error, sizeof(double));
	hipMalloc((double**)&d_abs_error, sizeof(double));

	int blocks = (ceil((double)(h_Px*h_Py)/1000));
	int threadsperblock = 1000;
	printf("Running on %d blocks each with %d threads\n",blocks,threadsperblock);

	prepare_grids<<<blocks, threadsperblock>>>(d_T,d_T_tmp,d_S,d_errors,d_grid_size,d_internal_size,d_Px,d_Py);
	hipDeviceSynchronize();
	int iter = 0;
	double h_conv_error = THRESH+1;
	//double h_abs_error = 0;
	while(h_conv_error > THRESH) {
		update_temporary<<<blocks,threadsperblock>>>(d_T,d_T_tmp,d_S,d_errors,d_grid_size,d_Px,d_Py);
		hipDeviceSynchronize();
		if(iter%1000==0) {
			//get_error<<<blocks,threadsperblock>>>(d_T,d_T_tmp,d_grid_size,d_Px,d_Py,d_conv_error);
			//get_abs_error<<<blocks,threadsperblock>>>(d_T,d_S,d_grid_size,d_abs_error);
			//cudaDeviceSynchronize();
			//cudaMemcpy(&h_conv_error, d_conv_error, sizeof(double), cudaMemcpyDeviceToHost);
			//cudaMemcpy(&h_abs_error, d_abs_error, sizeof(double), cudaMemcpyDeviceToHost);
			hipMemcpy(h_T, d_T, h_grid_size*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(h_T_tmp, d_T_tmp, h_grid_size*sizeof(double), hipMemcpyDeviceToHost);
			h_conv_error = 0;
			for(long i=0; i<h_grid_size; i++) {
				double val = fabs(h_T_tmp[i] - h_T[i]);
				if(val > h_conv_error)
					h_conv_error = val;
			}		
			//h_abs_error = powf(h_abs_error, 0.333);
			//h_conv_error = powf(h_conv_error, 0.333);
			printf("iter = %d... conv Error = %.10e\n", iter, h_conv_error);
		}
		update_real<<<blocks,threadsperblock>>>(d_T,d_T_tmp,d_S,d_errors,d_grid_size,d_Px,d_Py);
		iter++;
	}
	printf("Finished\n");
	
	hipMemcpy(h_T, d_T, h_grid_size*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_S, d_S, h_grid_size*sizeof(double), hipMemcpyDeviceToHost);
	double max = 0;
	for(long i=0; i<h_grid_size; i++) {
		double val = fabs(h_S[i] - h_T[i]);
		if(val > max)
			max = val;
	}
	printf("ABSLOUTE ERROR = %.10e\n", max);
	// Output .vtk file for ParaView
	double Xmin = 0;
	double Ymin = 0;
	double Xmax = XRANGE;
	double Ymax = YRANGE;
	VTK_out(h_Px, h_Py, &Xmin, &Xmax, &Ymin, &Ymax, h_T, 0);
	
}
